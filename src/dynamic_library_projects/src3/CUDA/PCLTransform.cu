#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

//#include <pcl/point_types.h>

struct PointXYZ{
	float x,y,z,u;
};

//~ struct Mat_Structure{
	//~ float _11_, _21_, _31_, _41_;
	//~ float _12_, _22_, _32_, _42_;
	//~ float _13_, _23_, _33_, _43_;
	//~ float _14_, _24_, _34_, _44_;
//~ };

__global__
void kernel(float *vec, float *mat, float *out, int problem_size){
    int tid = blockDim.x * blockIdx.x + threadIdx.x; //threadIdx.x;
	
	if(tid < problem_size)
	{
		#pragma unroll
		for(int i = 0; i<4; i++)
		{
			out[4*tid+i] = 0;
		}
		
		#pragma unroll
		for(int i=0; i<4; i++)
		{
			#pragma unroll
			for(int j=0; j<4; j++)
			{
				out[4*tid+i] += vec[4*tid+j] * mat[4*i+j];
			}
		}
	}
}


__global__
void transform_kernel(PointXYZ *vec, float *mat, int problem_size){
    int tid = blockDim.x * blockIdx.x + threadIdx.x; //threadIdx.x;
	
	if(tid < problem_size)
	{
		float x = 0;
		float y = 0;
		float z = 0;
		
		x = vec[tid].x * mat[0] + vec[tid].y * mat[1] + vec[tid].z * mat[2] + mat[3];
		y = vec[tid].x * mat[4] + vec[tid].y * mat[5] + vec[tid].z * mat[6] + mat[7];
		z = vec[tid].x * mat[8] + vec[tid].y * mat[9] + vec[tid].z * mat[10] + mat[11];
		
		vec[tid].x = x;
		vec[tid].y = y;
		vec[tid].z = z;
	}
}

//~ __global__
//~ void transform_kernel2(PointXYZ *vec, Mat_Structure mat, int problem_size){
    //~ int tid = blockDim.x * blockIdx.x + threadIdx.x; //threadIdx.x;
	
	//~ if(tid < problem_size)
	//~ {
		//~ float x = 0;
		//~ float y = 0;
		//~ float z = 0;
		
		//~ x = vec[tid].x * mat._11_ + vec[tid].y * mat._12_ + vec[tid].z * mat._13_ + mat._14_;
		//~ y = vec[tid].x * mat._21_ + vec[tid].y * mat._22_ + vec[tid].z * mat._23_ + mat._24_;
		//~ z = vec[tid].x * mat._31_ + vec[tid].y * mat._32_ + vec[tid].z * mat._33_ + mat._34_;
		
		//~ vec[tid].x = x;
		//~ vec[tid].y = y;
		//~ vec[tid].z = z;
	//~ }
//~ }


int testmain(float* vector_array, float* result_array, const int array_size, float* mat_4x4) {
	float *dev_array, *dev_mat, *dev_result;

	hipMalloc((void**)&dev_array, sizeof(float)*4*array_size);
	hipMalloc((void**)&dev_mat, sizeof(float)*16);
	hipMalloc((void**)&dev_result, sizeof(float)*4*array_size);

	hipMemcpy(dev_array, vector_array, sizeof(float)*4*array_size, hipMemcpyHostToDevice);
	hipMemcpy(dev_mat, mat_4x4, sizeof(float)*16, hipMemcpyHostToDevice);

    printf("\n\nRunning Kernel...\n\n");
	//kernel<<<1, array_size>>>(dev_array, dev_mat, dev_result);
	// Invoke kernel
	int threadsPerBlock = 1024;
	int blocksPerGrid = (array_size + threadsPerBlock - 1) / threadsPerBlock;
	kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_array, dev_mat, dev_result, array_size);

	hipMemcpy(result_array, dev_result, sizeof(float)*4*array_size, hipMemcpyDeviceToHost);

	hipFree(dev_array);
	hipFree(dev_mat);
	hipFree(dev_result);

	return 0;
};


int point_transform(PointXYZ * points, const int array_size, float* mat_4x4) {
	PointXYZ *dev_array;
	float *dev_mat;

	hipMalloc((void**)&dev_array, sizeof(PointXYZ)*array_size);
	hipMalloc((void**)&dev_mat, sizeof(float)*16);

	hipMemcpy(dev_array, points, sizeof(PointXYZ)*array_size, hipMemcpyHostToDevice);
	hipMemcpy(dev_mat, mat_4x4, sizeof(float)*16, hipMemcpyHostToDevice);

	// Invoke kernel
	int threadsPerBlock = 1024;
	int blocksPerGrid = (array_size + threadsPerBlock - 1) / threadsPerBlock;
	transform_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_array, dev_mat, array_size);

	hipMemcpy(points, dev_array, sizeof(PointXYZ)*array_size, hipMemcpyDeviceToHost);

	hipFree(dev_array);
	hipFree(dev_mat);

	return 0;
};

