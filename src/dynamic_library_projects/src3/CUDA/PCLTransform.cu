#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include <iostream>

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
      {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

//#include <pcl/point_types.h>

typedef struct{
	float x,y,z,u;
}PointXYZ;

//~ struct Mat_Structure{
	//~ float _11_, _21_, _31_, _41_;
	//~ float _12_, _22_, _32_, _42_;
	//~ float _13_, _23_, _33_, _43_;
	//~ float _14_, _24_, _34_, _44_;
//~ };

__global__
void kernel(float *vec, float *mat, float *out, int problem_size){
    int tid = blockDim.x * blockIdx.x + threadIdx.x; //threadIdx.x;
	
	if(tid < problem_size)
	{
		#pragma unroll
		for(int i = 0; i<4; i++)
		{
			out[4*tid+i] = 0;
		}
		
		#pragma unroll
		for(int i=0; i<4; i++)
		{
			#pragma unroll
			for(int j=0; j<4; j++)
			{
				out[4*tid+i] += vec[4*tid+j] * mat[4*i+j];
			}
		}
	}
}


__global__
void transform_kernel(PointXYZ *vec, float *mat, int problem_size){
    int tid = blockDim.x * blockIdx.x + threadIdx.x; //threadIdx.x;
	
	if(tid < problem_size)
	{
		float x = 0;
		float y = 0;
		float z = 0;
		
		x = vec[tid].x * mat[0] + vec[tid].y * mat[1] + vec[tid].z * mat[2] + mat[3];
		y = vec[tid].x * mat[4] + vec[tid].y * mat[5] + vec[tid].z * mat[6] + mat[7];
		z = vec[tid].x * mat[8] + vec[tid].y * mat[9] + vec[tid].z * mat[10] + mat[11];
		
		vec[tid].x = x;
		vec[tid].y = y;
		vec[tid].z = z;
	}
}


int testmain(float* vector_array, float* result_array, const int array_size, float* mat_4x4) {
	float *dev_array, *dev_mat, *dev_result;

	hipMalloc((void**)&dev_array, sizeof(float)*4*array_size);
	hipMalloc((void**)&dev_mat, sizeof(float)*16);
	hipMalloc((void**)&dev_result, sizeof(float)*4*array_size);

	hipMemcpy(dev_array, vector_array, sizeof(float)*4*array_size, hipMemcpyHostToDevice);
	hipMemcpy(dev_mat, mat_4x4, sizeof(float)*16, hipMemcpyHostToDevice);

    //~ printf("\n\nRunning Kernel...\n\n");
	//kernel<<<1, array_size>>>(dev_array, dev_mat, dev_result);
	// Invoke kernel
	int threadsPerBlock = 1024;
	int blocksPerGrid = (array_size + threadsPerBlock - 1) / threadsPerBlock;
	kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_array, dev_mat, dev_result, array_size);

	hipMemcpy(result_array, dev_result, sizeof(float)*4*array_size, hipMemcpyDeviceToHost);

	hipFree(dev_array);
	hipFree(dev_mat);
	hipFree(dev_result);

	return 0;
};


int point_transform(PointXYZ * points, const int array_size, float* mat_4x4) {
	
	//~ printf("\n\nRunning Kernel...\n\n");
	
	PointXYZ *dev_array;
	float *dev_mat;

	HANDLE_ERROR(hipMalloc(&dev_array, sizeof(PointXYZ)*array_size));
	HANDLE_ERROR(hipMalloc(&dev_mat, sizeof(float)*16));
	
	//~ cudaMalloc((void**)&dev_array, sizeof(PointXYZ)*array_size);
	//~ cudaMalloc((void**)&dev_mat, sizeof(float)*16);

	//~ std::cout << "cudalib: (" << points[0].x << "," << points[0].y << "," << points[0].z << ")" << std::endl;

	HANDLE_ERROR(hipMemcpy(dev_array, points, sizeof(PointXYZ)*array_size, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_mat, mat_4x4, sizeof(float)*16, hipMemcpyHostToDevice));

	// Invoke kernel
	//~ int threadsPerBlock = 128;
	//~ int blocksPerGrid = (array_size + threadsPerBlock - 1) / threadsPerBlock;
	//~ transform_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_array, dev_mat, array_size);
	transform_kernel<<<16, 128>>>(dev_array, dev_mat, array_size);

	HANDLE_ERROR(hipMemcpy(points, dev_array, sizeof(PointXYZ)*array_size, hipMemcpyDeviceToHost));

	//~ std::cout << "cudalib after: (" << points[0].x << "," << points[0].y << "," << points[0].z << ")" << std::endl;

	hipFree(dev_array);
	hipFree(dev_mat);

	return 0;
};

int point_transform3(PointXYZ * points, const int array_size, PointXYZ * result_points, float* mat_4x4) {
	PointXYZ *dev_array;
	float *dev_mat;

	hipMalloc((void**)&dev_array, sizeof(PointXYZ)*array_size);
	hipMalloc((void**)&dev_mat, sizeof(float)*16);

	hipMemcpy(dev_array, points, sizeof(PointXYZ)*array_size, hipMemcpyHostToDevice);
	hipMemcpy(dev_mat, mat_4x4, sizeof(float)*16, hipMemcpyHostToDevice);

	// Invoke kernel
	int threadsPerBlock = 1024;
	int blocksPerGrid = (array_size + threadsPerBlock - 1) / threadsPerBlock;
	transform_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_array, dev_mat, array_size);

	hipMemcpy(result_points, dev_array, sizeof(PointXYZ)*array_size, hipMemcpyDeviceToHost);

	hipFree(dev_array);
	hipFree(dev_mat);

	return 0;
};

